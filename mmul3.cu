#include "hip/hip_runtime.h"
/*
N*N行列の行列積。共有メモリを用いる一般的な方法。
1ブロックが1列を担当。1スレッドは1要素を担当、スレッド数が足りなければその分ループ。

行列ABのBの計算する列を共有メモリに移して実行。
*/

#include "book.h"
#include <hipblas.h>
#include <hiprand.h>
#include <sys/time.h>

#define N 16384
#define Block 8192
#define Thread 1024
#define BLOCK_SIZE 32

void GPU_fill_rand(float *a, int row, int col){
  hiprandGenerator_t prng;
  hiprandCreateGenerator(&prng, HIPRAND_RNG_PSEUDO_DEFAULT);

  hiprandSetPseudoRandomGeneratorSeed(prng, (unsigned long long) clock());

  hiprandGenerateUniform(prng, a, row*col);
}

void GPU_blas_mmul(const float *a, const float *b, float *c, const int m, const int k, const int n){
  float *dev_a, *dev_b, *dev_c;
  float alf = 1.0;
  float bet = 0.0;
  float *alpha = &alf;
  float *beta = &bet;
  HANDLE_ERROR(hipMalloc((void**)&dev_a, N*N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N*N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N*N*sizeof(float)));
  HANDLE_ERROR(hipMemcpy(dev_a, a, N*N*sizeof(float), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N*N*sizeof(float), hipMemcpyHostToDevice));

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha, dev_a, N, dev_b, N, beta, dev_c, N);
  HANDLE_ERROR(hipMemcpy(c, dev_c, N*N*sizeof(float), hipMemcpyDeviceToHost));

  hipblasDestroy(handle);

  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
}

__global__ void mmul(float *a, float *b, float *c){
  long tid = blockIdx.x * Thread + threadIdx.x;
  while(tid < N * N){
    int row = tid / N;
    int col = tid % N;
    c[tid] = 0;
    for(int i = 0; i < N; i++){
      c[row*N+col] += a[row*N+i] * b[i*N+col];
    }
    tid += Thread * Block;
  }
}

__global__ void mmul3(float *a, float *b, float *c, int n){
  int row = threadIdx.y, col = threadIdx.x;
  int arow = blockIdx.y*BLOCK_SIZE + row;
  int bcol = blockIdx.x*BLOCK_SIZE + col;

  __shared__ float cachea[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float cacheb[BLOCK_SIZE][BLOCK_SIZE];
  float sum = 0.0;

  int ax = arow*n+col;
  int bx = row*n+bcol;
  for(int i = 0; i < n; ){
    cachea[row][col] = a[ax+i];
    cacheb[row][col] = b[bx+i*n];
    i += BLOCK_SIZE;
    __syncthreads();

    #pragma unroll
    for(int j = 0; j < BLOCK_SIZE; j++){
      sum += cachea[row][j]*cacheb[j][col];
    }
    __syncthreads();
  }
  c[arow*N+bcol] = sum;
}

__global__ void transpose(float *a){                            //OK
  long tid = blockIdx.x * Thread + threadIdx.x;
  while(tid < N * N){
    int row = tid / N;
    int col = tid % N;
    if(row < col){
      float x = a[row*N+col];
      a[row*N+col] = a[col*N+row];
      a[col*N+row] = x;
    }
    tid += Thread * Block;
  }
}

void mmulcpu(float *a, float *b, float *c){
  for(int i = 0; i < N; i++){
    for(int j = 0; j < N; j++){
      c[i*N+j] = 0.0;
      for(int k = 0; k < N; k++){
        c[i*N+j] += a[i*N+k]*b[k*N+j];
      }
    }
  }
}

void printm(float *a, int n, int m){
  for(int i = 0; i < n; i++){
    for(int j = 0; j < m; j++){
      printf("%f ", a[i*N+j]);
    }
    printf("\n");
  }
  printf("\n");
}

int main(void){
  float *a, *b, *c1, *c2, *c3, *c4;
  float *dev_a, *dev_b, *dev_c, *dev_c2, *dev_c3;
  struct timeval t0, t1;

  a = (float *)malloc(N*N*sizeof(float));
  b = (float *)malloc(N*N*sizeof(float));
  c1 = (float *)malloc(N*N*sizeof(float));
  c2 = (float *)malloc(N*N*sizeof(float));
  c3 = (float *)malloc(N*N*sizeof(float));
  c4 = (float *)malloc(N*N*sizeof(float));

  HANDLE_ERROR(hipMalloc((void**)&dev_a, N*N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N*N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N*N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c2, N*N*sizeof(float)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c3, N*N*sizeof(float)));
  HANDLE_ERROR(hipMemset(dev_c, 0, N*N*sizeof(float)));
  HANDLE_ERROR(hipMemset(dev_c2, 0, N*N*sizeof(float)));
  HANDLE_ERROR(hipMemset(dev_c3, 0, N*N*sizeof(float)));

  GPU_fill_rand(dev_a, N, N);
  GPU_fill_rand(dev_b, N, N);

  HANDLE_ERROR(hipMemcpy(a, dev_a, N*N*sizeof(float), hipMemcpyDeviceToHost));
  HANDLE_ERROR(hipMemcpy(b, dev_b, N*N*sizeof(float), hipMemcpyDeviceToHost));
  //mmulcpu(a, b, c4);

  gettimeofday(&t0, NULL);
  //mmul<<<Block, Thread>>>(dev_a, dev_b, dev_c);
  hipDeviceSynchronize();
  gettimeofday(&t1, NULL);
  printf("Elapsed time(mmul)= %.10lf\n", (double)(t1.tv_sec - t0.tv_sec) + (double)(t1.tv_usec - t0.tv_usec)/(1000.0*1000.0));
  HANDLE_ERROR(hipMemcpy(c1, dev_c, N*N*sizeof(float), hipMemcpyDeviceToHost));

  /*
  gettimeofday(&t0, NULL);
  mmul2<<<Block, Thread>>>(dev_a, dev_b, dev_c2);
  hipDeviceSynchronize();
  gettimeofday(&t1, NULL);
  printf("Elapsed time(mmul2)= %.10lf\n", (double)(t1.tv_sec - t0.tv_sec) + (double)(t1.tv_usec - t0.tv_usec)/(1000.0*1000.0));
  HANDLE_ERROR(hipMemcpy(c2, dev_c2, N*N*sizeof(float), hipMemcpyDeviceToHost));
*/

  gettimeofday(&t0, NULL);
  dim3 block(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(N/BLOCK_SIZE, N/BLOCK_SIZE);
  mmul3<<<grid, block>>>(dev_a, dev_b, dev_c2, N);
  hipDeviceSynchronize();
  gettimeofday(&t1, NULL);
  printf("Elapsed time(mmul2)= %.10lf\n", (double)(t1.tv_sec - t0.tv_sec) + (double)(t1.tv_usec - t0.tv_usec)/(1000.0*1000.0));
  HANDLE_ERROR(hipMemcpy(c2, dev_c2, N*N*sizeof(float), hipMemcpyDeviceToHost));

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  float alpha = 1.0, beta = 0.0;

  gettimeofday(&t0, NULL);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, dev_b, N, dev_a, N, &beta, dev_c3, N);
  hipDeviceSynchronize();
  gettimeofday(&t1, NULL);
  printf("Elapsed time(cublas)= %.10lf\n", (double)(t1.tv_sec - t0.tv_sec) + (double)(t1.tv_usec - t0.tv_usec)/(1000.0*1000.0));

  //transpose<<<Block, Thread>>>(dev_c3);

  HANDLE_ERROR(hipMemcpy(c3, dev_c3, N*N*sizeof(float), hipMemcpyDeviceToHost));
  hipblasDestroy(handle);

  //transpose<<<Block, Thread>>>(dev_a);
  //transpose<<<Block, Thread>>>(dev_b);

  int wrong1 = 0, wrong2 = 0, wrong3 = 0, wrong4 = 0;
  for(int i = 0; i < N; i++){
    for(int j = 0; j < N; j++){
      if(c1[i*N+j] != c2[i*N+j]){
        wrong1++;
      }
      if(c1[i*N+j] != c3[i*N+j]) wrong2++;
      if(c1[i*N+j] != c4[i*N+j]) wrong4++;
      if(c2[i*N+j] != c3[i*N+j]){
        wrong3++;
        if(wrong3 < 20) printf("c2:%f, c3:%f\n", c1[i*N+j], c3[i*N+j]);
      }
    }
  }

  //printm(c, N, N);
  //printm(c2, N, N);
  //printm(c3, N, N);
  //printm(c4, N, N);
  for(int i = 0; i < 10; i++){
    for(int j = 0; j < 10; j++){
      //printf("%f %f\n", c1[i*N+j], c2[i*N+j]);
    }
  }
  printf("wrong1 : %d\n", wrong1);
  printf("wrong2 : %d\n", wrong2);
  printf("wrong3 : %d\n", wrong3);
  //printf("wrong4 : %d\n", wrong4)
  //printm(c2, N, N);

  free(a);
  free(b);
  free(c1);
  free(c2);
  free(c3);
  free(c4);

  HANDLE_ERROR(hipFree(dev_a));
  HANDLE_ERROR(hipFree(dev_b));
  HANDLE_ERROR(hipFree(dev_c));
  HANDLE_ERROR(hipFree(dev_c2));
  HANDLE_ERROR(hipFree(dev_c3));


  return 0;
}
